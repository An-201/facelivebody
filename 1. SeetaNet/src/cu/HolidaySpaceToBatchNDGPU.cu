#include "hip/hip_runtime.h"
#include"HolidaySpaceToBatchNDGPU.h"
#include"HolidayCommonCuda.h"

HolidaySpaceToBatchNDGPU::HolidaySpaceToBatchNDGPU()
{

}

HolidaySpaceToBatchNDGPU::~HolidaySpaceToBatchNDGPU()
{

}

void HolidaySpaceToBatchNDGPU::CaculateOutputSize(int input_number, int input_height, int input_width, int input_channels,
													 int &output_number, int &output_height, int &output_width, int &output_channels)
{
	output_number = input_number * m_block_shape[0] * m_block_shape[1];
	output_height = (input_height + m_paddings[0] + m_paddings[1]) / m_block_shape[0];
	output_width = (input_width + m_paddings[2] + m_paddings[3]) / m_block_shape[1];
	output_channels = input_channels;
}

int HolidaySpaceToBatchNDGPU::Init(Holiday_LayerParameter& inputparam, HolidayNetResource<float> *pNetResource)
{
    pNetResourceGpu = (HolidayNetResourceGpu *)pNetResource->pNetResourceGpu;

	// set bottom size
	int bottom_index = inputparam.bottom_index(0);
	HolidayDataSize bottom_size = pNetResource->feature_vector_size[bottom_index];
	this->bottom_data_size.resize(1);
	this->bottom_data_size[0] = bottom_size;

	// read param from inputparam.spacetobatchnd_param();
	auto &param = inputparam.spacetobatchnd_param();
	for (int i = 0; i < param.block_shape_size(); i++)
	{
		m_block_shape.push_back(param.block_shape(i));
	}
	for (int i = 0; i < param.paddings_size(); i++)
	{
		m_paddings.push_back(param.paddings(i));
	}

	assert(m_block_shape.size() == 2 && m_paddings.size() == 4);

    CUDA_ERROR(SafeCudaMalloc((void **)&m_block_shape_gpu, m_block_shape.size() * sizeof(int)));
    CUDA_ERROR(SafeCudaMalloc((void **)&m_paddings_gpu, m_paddings.size() * sizeof(int)));
    CUDA_ERROR(hipMemcpyAsync(m_block_shape_gpu, m_block_shape.data(), m_block_shape.size() * sizeof(int), hipMemcpyHostToDevice, pNetResourceGpu->main_stream));
    CUDA_ERROR(hipMemcpyAsync(m_paddings_gpu, m_paddings.data(), m_paddings.size() * sizeof(int), hipMemcpyHostToDevice, pNetResourceGpu->main_stream));

	// set top size
	this->top_data_size.resize(1);
	this->top_data_size[0].data_dim.resize(4);
	CaculateOutputSize(this->bottom_data_size[0].data_dim, this->top_data_size[0].data_dim);

	hipStreamSynchronize(pNetResourceGpu->main_stream);
    return CUDA_RETURN_VALUE;
}

int HolidaySpaceToBatchNDGPU::Exit() {
    if (m_block_shape_gpu) hipFree(m_block_shape_gpu);
    if (m_paddings_gpu) hipFree(m_paddings_gpu);

    return CUDA_RETURN_VALUE;
}

// number, channels, height, width means
__global__ static void gSpaceToBatchND_kernel(float *input_data, float *output_data,
											  int input_number, int input_channels, int input_height, int input_width,
											  int output_number, int output_channels, int output_height, int output_width,
											  int input_size, int input_number_step, int input_channels_step, int input_height_step,
											  int output_size, int output_number_step, int output_channels_step, int output_height_step,
											  int *block_shape, int *paddings)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	auto &B = block_shape;
	auto &P = paddings;

	if (index < input_size)
	{
		int at_input_i = index;

		int n = index / input_number_step;
		index %= input_number_step;
		int c = index / input_channels_step;
		index %= input_channels_step;
		int h = index / input_height_step;
		index %= input_height_step;
		int w = index;

		int on = ((h + P[0]) % B[0] * B[1] + (w + P[2]) % B[1]) * input_number + n;
		int oc = c;
		int oh = (h + P[0]) / B[0];
		int ow = (w + P[2]) / B[1];

		int at_output_i = on * output_number_step
						  + oc * output_channels_step
						  + oh * output_height_step
						  + ow;

		output_data[at_output_i] = input_data[at_input_i];
	}
}

int HolidaySpaceToBatchNDGPU::Process(std::vector<HolidayFeatureMap<float>*> input_data_map, std::vector<HolidayFeatureMap<float>*>& output_data_map)
{
    // trans param to gpu
	input_data_map[0]->m_gpu.shape_ = input_data_map[0]->data_shape;
	input_data_map[0]->m_gpu.Gpu_DataIn(pNetResourceGpu, input_data_map[0]->dwStorageType, input_data_map[0]->m_cpu.dataMemoryPtr());
	input_data_map[0]->dwStorageType = DATA_GPU;
	output_data_map[0]->dwStorageType = DATA_GPU;

	// set output data type and shape

	// set output data shape
	CaculateOutputSize(input_data_map[0]->data_shape, output_data_map[0]->data_shape);

	// set output gpu shape
	output_data_map[0]->m_gpu.shape_ = output_data_map[0]->data_shape;
	output_data_map[0]->m_gpu.data_size = output_data_map[0]->data_shape[0] * output_data_map[0]->data_shape[1] * output_data_map[0]->data_shape[2] * output_data_map[0]->data_shape[3];


	// write output
	int input_number = input_data_map[0]->data_shape[0];
	int input_channels = input_data_map[0]->data_shape[1];
	int input_height = input_data_map[0]->data_shape[2];
	int input_width = input_data_map[0]->data_shape[3];

	int output_number = output_data_map[0]->data_shape[0];
	int output_channels = output_data_map[0]->data_shape[1];
	int output_height = output_data_map[0]->data_shape[2];
	int output_width = output_data_map[0]->data_shape[3];

	int input_size = input_number * input_channels * input_height * input_width;
	int input_number_step = input_channels * input_height * input_width;
	int input_channels_step = input_height * input_width;
	int input_height_step = input_width;

	int output_size = output_number * output_channels * output_height * output_width;
	int output_number_step = output_channels * output_height * output_width;
	int output_channels_step = output_height * output_width;
	int output_height_step = output_width;

	float *input_data = input_data_map[0]->m_gpu.pfData_gpu;
	float *output_data = output_data_map[0]->m_gpu.pfData_gpu;

	hipMemsetAsync(output_data, 0, output_size * sizeof(float), pNetResourceGpu->main_stream);

	gSpaceToBatchND_kernel << <CUDA_BLOCK(input_size, CUDA_THREAD_NUM), CUDA_THREAD_NUM, 0, pNetResourceGpu->main_stream>> >
			(input_data, output_data,
					input_number, input_channels, input_height, input_width,
					output_number, output_channels, output_height, output_width,
					input_size, input_number_step, input_channels_step, input_height_step,
					output_size, output_number_step, output_channels_step, output_height_step,
					m_block_shape_gpu, m_paddings_gpu);

	return CUDA_RETURN_VALUE;
}
