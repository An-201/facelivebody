#include "hip/hip_runtime.h"
#include"HolidayPowerGPU.h"

HolidayPowerGPU::HolidayPowerGPU()
{
}
HolidayPowerGPU::~HolidayPowerGPU()
{
}
__global__ static void gPower_kernel(float *pfDataIn, float *pfDataOut, int dwSize, float fScale, float fShift, float fPower)
{
	int dwIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (dwIdx < dwSize)
	{
		pfDataOut[dwIdx] = powf(fScale * pfDataIn[dwIdx] + fShift, fPower);
	}
}

int HolidayPowerGPU::Init(Holiday_LayerParameter& inputparam, HolidayNetResource<float> *pNetResource)
{
	pNetResourceGpu = (HolidayNetResourceGpu *)pNetResource->pNetResourceGpu;
	
	//bottom_data_size = inputparam.bottom_data_size;
	int index = inputparam.bottom_index(0);
	bottom_data_size.resize(1);
	bottom_data_size[0] = pNetResource->feature_vector_size[index];

	top_data_size = bottom_data_size;
	fScale = inputparam.power_param().scale();
	fShift = inputparam.power_param().shift();
	fPower = inputparam.power_param().power();
	return CUDA_RETURN_VALUE;
}
int HolidayPowerGPU::Exit()
{
	return CUDA_RETURN_VALUE;
}
//int Power_gpu::GetTopSize(std::vector<DataSize>& out_data_size)
//{
//	out_data_size = top_data_size;
//	return CUDA_RETURN_VALUE;
//}
int HolidayPowerGPU::Process(std::vector<HolidayFeatureMap<float>*> input_data_map, std::vector<HolidayFeatureMap<float>*>& output_data_map)
{
#ifdef _DEBUG
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
#endif
	input_data_map[0]->m_gpu.shape_ = input_data_map[0]->data_shape;
	input_data_map[0]->m_gpu.Gpu_DataIn(pNetResourceGpu, input_data_map[0]->dwStorageType, input_data_map[0]->m_cpu.dataMemoryPtr());
	input_data_map[0]->dwStorageType = DATA_GPU;
	output_data_map[0]->dwStorageType = DATA_GPU;
	output_data_map[0]->data_shape = input_data_map[0]->data_shape;
	output_data_map[0]->m_gpu.shape_ = output_data_map[0]->data_shape;
	output_data_map[0]->m_gpu.data_size = input_data_map[0]->m_gpu.data_size;
	gPower_kernel << <CUDA_BLOCK(output_data_map[0]->m_gpu.data_size, CUDA_THREAD_NUM), CUDA_THREAD_NUM, 0, pNetResourceGpu->main_stream>> >
		((float *)input_data_map[0]->m_gpu.pfData_gpu, (float *)output_data_map[0]->m_gpu.pfData_gpu, output_data_map[0]->m_gpu.data_size, fScale, fShift, fPower);

#ifdef _DEBUG
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf(" Power: %f ms \n", msecTotal1);
#endif
#ifdef _DEBUG
	int dwSize = top_data_size[0].data_dim[1] * top_data_size[0].data_dim[2] * top_data_size[0].data_dim[3];
	float *pfDataOut = new float[dwSize];
	hipMemcpy(pfDataOut, output_data_map[0]->m_gpu.pfData_gpu,dwSize * sizeof(float), hipMemcpyDeviceToHost);
	delete[] pfDataOut;
	hipDeviceSynchronize();
	printf("Power:%s\n", hipGetErrorString(hipGetLastError()));
#endif
	return CUDA_RETURN_VALUE;
}
