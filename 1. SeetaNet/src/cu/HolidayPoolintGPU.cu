#include "hip/hip_runtime.h"
#include "HolidayPoolintGPU.h"
#include "HolidayCommonCuda.h"

enum HolidayPoolingParameter_PoolMethod {
	PoolingParameter_PoolMethod_MAX = 0,
	PoolingParameter_PoolMethod_AVE = 1,
	PoolingParameter_PoolMethod_STOCHASTIC = 2
};
Pooling_gpu::Pooling_gpu()
{
}
Pooling_gpu::~Pooling_gpu()
{
}
__global__ static void gMaxPooling_kernel(float  *pfDataIn, float *pfDataOut, int dwSize,
	int dwRowIn, int dwColIn, int dwSliceIn, int dwRowOut, int dwColOut, int dwSliceOut,
	int dwKernelH, int dwKernelW, int dwStrideH, int dwStrideW, int dwPadH, int dwPadW)
{
	int dwIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (dwIdx < dwSize)
	{
		int dwDimN = dwIdx / (dwSliceOut * dwRowOut * dwColOut);
		int dwDim2S = dwIdx % (dwSliceOut * dwRowOut * dwColOut) / (dwRowOut * dwColOut);
		int dwDim2R = dwIdx % (dwRowOut * dwColOut) / dwColOut;
		int dwDim2C = dwIdx % dwColOut;
		int dwDim1R = dwDim2R * dwStrideH - dwPadH;
		int dwDim1C = dwDim2C * dwStrideW - dwPadW;
		int dwDim1S = dwDim2S;
		float fMax;
		fMax = -3.402823E38;
		for (int i = Holliday_MAX(dwDim1R, 0); i < Holliday_MIN(dwDim1R + dwKernelH, dwRowIn); i++)
		{
			for (int j = Holliday_MAX(dwDim1C, 0); j < Holliday_MIN(dwDim1C + dwKernelW, dwColIn); ++j)
			{
				fMax = fmaxf(fMax, pfDataIn[j + dwColIn * (i + dwRowIn * (dwDim1S + dwSliceIn * dwDimN))]);
			}
		}
		pfDataOut[dwIdx] = fMax;
	}
}
__global__ static void gAveragePooling_kernel(float  *pfDataIn, float *pfDataOut, int dwSize,
	int dwRowIn, int dwColIn, int dwSliceIn, int dwRowOut, int dwColOut, int dwSliceOut,
	int dwKernelH, int dwKernelW, int dwStrideH, int dwStrideW, int dwPadH, int dwPadW)
{
	int dwIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (dwIdx < dwSize)
	{
		int dwDimN = dwIdx / (dwSliceOut * dwRowOut * dwColOut);
		int dwDim2S = dwIdx % (dwSliceOut * dwRowOut * dwColOut) / (dwRowOut * dwColOut);
		int dwDim2R = dwIdx % (dwRowOut * dwColOut) / dwColOut;
		int dwDim2C = dwIdx % dwColOut;
		int dwDim1R = dwDim2R * dwStrideH - dwPadH;
		int dwDim1C = dwDim2C * dwStrideW - dwPadW;
		int dwDim1S = dwDim2S;
		float fSum;
		int dwCount;
		fSum = 0.f;
		dwCount = 0;
		for (int i = Holliday_MAX(dwDim1R, 0); i < Holliday_MIN(dwDim1R + dwKernelH, dwRowIn); i++)
		{
			for (int j = Holliday_MAX(dwDim1C, 0); j < Holliday_MIN(dwDim1C + dwKernelW, dwColIn); ++j)
			{
				fSum += pfDataIn[j + dwColIn * (i + dwRowIn * (dwDim1S + dwSliceIn * dwDimN))];
				++dwCount;
			}
		}
		pfDataOut[dwIdx] = fSum / dwCount;
	}
}
int Pooling_gpu::Init(Holiday_LayerParameter& inputparam, HolidayNetResource<float> *pNetResource)
{
	pNetResourceGpu = (HolidayNetResourceGpu *)pNetResource->pNetResourceGpu;

	dwType = inputparam.pooling_param().pool();
	
	int index = inputparam.bottom_index(0);
	bottom_data_size.resize(1);
	bottom_data_size[0] = pNetResource->feature_vector_size[index];
	
	
	dwStrideH = inputparam.pooling_param().stride_height();
	dwStrideW = inputparam.pooling_param().stride_width();
	dwKernelH = inputparam.pooling_param().kernel_height();
	dwKernelW = inputparam.pooling_param().kernel_width();
	dwPadH = inputparam.pooling_param().pad_height();
	dwPadW = inputparam.pooling_param().pad_width();

	dwValid = false;
	if (inputparam.pooling_param().has_valid())
	{
		dwValid = inputparam.pooling_param().valid();
	}

	if (inputparam.pooling_param().global_pooling())
	{
		dwKernelH = bottom_data_size[0].data_dim[2];
		dwKernelW = bottom_data_size[0].data_dim[3];
		dwPadH = 0;
		dwPadW = 0;
	}
	top_data_size.resize(1);
	top_data_size[0].data_dim[1] = bottom_data_size[0].data_dim[1];
	if (dwValid)
	{
		top_data_size[0].data_dim[2] = floor((bottom_data_size[0].data_dim[2] - dwKernelH + 2 * dwPadH) / (float)dwStrideH + 1);
		top_data_size[0].data_dim[3] = floor((bottom_data_size[0].data_dim[3] - dwKernelW + 2 * dwPadW) / (float)dwStrideW + 1);
	}
	else
	{
		top_data_size[0].data_dim[2] = ceil((bottom_data_size[0].data_dim[2] - dwKernelH + 2 * dwPadH) / (float)dwStrideH + 1);
		top_data_size[0].data_dim[3] = ceil((bottom_data_size[0].data_dim[3] - dwKernelW + 2 * dwPadW) / (float)dwStrideW + 1);
	}
	//int dwSize = top_data_size[0].data_dim[2] * top_data_size[0].data_dim[3] * top_data_size[0].data_dim[1];

	return CUDA_RETURN_VALUE;
}
int Pooling_gpu::Exit()
{
	return CUDA_RETURN_VALUE;
}
int Pooling_gpu::GetTopSize(std::vector<HolidayDataSize>& out_data_size)
{
	out_data_size = top_data_size;
	return CUDA_RETURN_VALUE;
}
int Pooling_gpu::Process(std::vector<HolidayFeatureMap<float>*> input_data_map, std::vector<HolidayFeatureMap<float>*>& output_data_map)
{
#ifdef _DEBUG
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
#endif
	input_data_map[0]->m_gpu.Gpu_DataIn(pNetResourceGpu, input_data_map[0]->dwStorageType, input_data_map[0]->m_cpu.dataMemoryPtr());
	if (PoolingParameter_PoolMethod_AVE== dwType)
	{
		int dwSize = top_data_size[0].data_dim[1] * top_data_size[0].data_dim[2] * top_data_size[0].data_dim[3];
		gAveragePooling_kernel << <CUDA_BLOCK(dwSize * input_data_map[0]->m_gpu.shape_[0], CUDA_THREAD_NUM), CUDA_THREAD_NUM, 0, pNetResourceGpu->main_stream>> >
			(input_data_map[0]->m_gpu.pfData_gpu, output_data_map[0]->m_gpu.pfData_gpu,
		dwSize* input_data_map[0]->m_gpu.shape_[0],
		bottom_data_size[0].data_dim[2], bottom_data_size[0].data_dim[3], bottom_data_size[0].data_dim[1],
		top_data_size[0].data_dim[2], top_data_size[0].data_dim[3], top_data_size[0].data_dim[1],
		dwKernelH, dwKernelW, dwStrideH, dwStrideW, dwPadH, dwPadW);
	}
	else if (PoolingParameter_PoolMethod_MAX == dwType)
	{
		int dwSize = top_data_size[0].data_dim[1] * top_data_size[0].data_dim[2] * top_data_size[0].data_dim[3];
		gMaxPooling_kernel << <CUDA_BLOCK(dwSize * input_data_map[0]->m_gpu.shape_[0], CUDA_THREAD_NUM), CUDA_THREAD_NUM, 0, pNetResourceGpu->main_stream>> >
			(input_data_map[0]->m_gpu.pfData_gpu, output_data_map[0]->m_gpu.pfData_gpu,
			dwSize * input_data_map[0]->m_gpu.shape_[0],
			bottom_data_size[0].data_dim[2], bottom_data_size[0].data_dim[3], bottom_data_size[0].data_dim[1],
			top_data_size[0].data_dim[2], top_data_size[0].data_dim[3], top_data_size[0].data_dim[1],
			dwKernelH, dwKernelW, dwStrideH, dwStrideW, dwPadH, dwPadW);
	}
	else if (PoolingParameter_PoolMethod_STOCHASTIC == dwType)
	{
	}
	else
	{
	}
	output_data_map[0]->dwStorageType = DATA_GPU;
	output_data_map[0]->data_shape[0] = input_data_map[0]->data_shape[0];
	output_data_map[0]->data_shape[1] = top_data_size[0].data_dim[1];
	output_data_map[0]->data_shape[2] = top_data_size[0].data_dim[2];
	output_data_map[0]->data_shape[3] = top_data_size[0].data_dim[3];
	return CUDA_RETURN_VALUE;
}